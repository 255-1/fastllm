#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "fastllm-cuda.h"
#include "fastllm.h"

static hipblasHandle_t fastllmCublasHandle = nullptr;

void FastllmMatMulInt8(int8_t *A, int8_t *B, int32_t *C, int n, int m, int k) {
    int32_t i_alpha = 1, i_beta = 0;
    if (fastllmCublasHandle == nullptr) {
        hipblasCreate(&fastllmCublasHandle);
    }

    hipDeviceSynchronize();
    hipDataType AType = HIP_R_8I, BType = HIP_R_8I, CType = HIP_R_32I, ComputeType = HIP_R_32I;
    hipblasStatus_t status;
    status = hipblasGemmEx(fastllmCublasHandle,
                          HIPBLAS_OP_T,
                          HIPBLAS_OP_N,
                          n,
                          k,
                          m,
                          &i_alpha,
                          A,
                          AType,
                          m,
                          B,
                          BType,
                          m,
                          &i_beta,
                          C,
                          CType,
                          n,
                          ComputeType,
                          static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Error: cublas error.\n");
        exit(0);
    }
    hipDeviceSynchronize();
}


__global__ void MatMulFloatInt8Kernel(float *A, uint8_t *B, float *C, float *bias, float *scales, uint8_t *zeros,
                                      int n, int m, int k) {
    int idx = blockIdx.x;
    int idy = threadIdx.x;
    int curId = idx * 64 + idy;
    int per = n * k / 4096;

    int st = curId * per, end = st + per;
    if (curId == 4095) {
        end = n * k;
    }

    for (int id = st; id < end; id++) {
        int i = id / k;
        int j = id % k;
        float now = 0.0f;
        int l = 0;
        for (; l < m; l++) {
            now += A[i * m + l] * (B[j * m + l] - zeros[j]);
        }

        now = now * scales[j];
        now += bias[j];
        C[i * k + j] = now;
    }
}


bool FastllmMatMulFloatInt8(const fastllm::Data &input, fastllm::Data &weight, const fastllm::Data &bias, fastllm::Data &output, int n, int m, int k) {
    float *inputData = (float *) input.cpuData;
    uint8_t *weightData = (uint8_t *) weight.cpuData;
    float *outputData = (float *) output.cpuData;
    float *biasData = bias.dims.size() > 0 ? (float *) bias.cpuData : nullptr;

    float *cudaScales;
    uint8_t *cudaZeropoints;
    float *cudaBiasData;

    hipMalloc(&cudaScales, k * sizeof(float));
    hipMalloc(&cudaZeropoints, k);
    hipMalloc(&cudaBiasData, k * sizeof(float));

    float *scales = new float[k];
    uint8_t *zeropoints = new uint8_t[k];
    float *biass = new float[k];
    for (int i = 0; i < k; i++) {
        zeropoints[i] = weight.perChannelsConfigs[i].zeroPoint;
        scales[i] = weight.perChannelsConfigs[i].scale;
        biass[i] = (biasData ? biasData[i] : 0.0f);
    }

    hipMemcpy(cudaScales, scales, k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaZeropoints, zeropoints, k, hipMemcpyHostToDevice);
    hipMemcpy(cudaBiasData, biass, k * sizeof(float), hipMemcpyHostToDevice);
/*
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            float now = 0.0f;
            int l = 0;
            for (; l < m; l++) {
                now += inputData[i * m + l] * (weightData[j * m + l] - zeropoints[j]);
            }

            now = now * scales[j];
            now += biass[j];
            outputData[i * k + j] = now;
        }
    }
*/
    float *cudaOutput, *cudaInput;
    hipMalloc(&cudaInput, n * m * sizeof(float));
    hipMalloc(&cudaOutput, n * k * sizeof(float));
    hipMemcpy(cudaInput, inputData, n * m * sizeof(float), hipMemcpyHostToDevice);

    MatMulFloatInt8Kernel <<< 64, 64 >>> (cudaInput, (uint8_t*)weight.cudaData, cudaOutput,
                                                     cudaBiasData, cudaScales, cudaZeropoints, n, m, k);

    hipMemcpy(outputData, cudaOutput, n * k * sizeof(float), hipMemcpyDeviceToHost);

    delete[] zeropoints;
    delete[] scales;
    delete[] biass;
    hipFree(cudaZeropoints);
    hipFree(cudaScales);
    hipFree(cudaBiasData);
    hipFree(cudaOutput);

    return true;
}

void * FastllmCudaMalloc(size_t size) {
    void * ret;
    hipMalloc(&ret, size);
    return ret;
}

void FastllmCudaCopyFromHostToDevice(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void FastllmCudaCopyFromDeviceToHost(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void FastllmCudaFree(void *ret) {
    hipFree(ret);
}