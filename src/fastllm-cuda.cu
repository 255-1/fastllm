#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "fastllm-cuda.h"
#include "fastllm.h"

static hipblasHandle_t fastllmCublasHandle = nullptr;

void FastllmMatMulInt8(int8_t *A, int8_t *B, int32_t *C, int n, int m, int k) {
    int32_t i_alpha = 1, i_beta = 0;
    if (fastllmCublasHandle == nullptr) {
        hipblasCreate(&fastllmCublasHandle);
    }

    hipDeviceSynchronize();
    hipDataType AType = HIP_R_8I, BType = HIP_R_8I, CType = HIP_R_32I, ComputeType = HIP_R_32I;
    hipblasStatus_t status;
    status = hipblasGemmEx(fastllmCublasHandle,
                          HIPBLAS_OP_T,
                          HIPBLAS_OP_N,
                          n,
                          k,
                          m,
                          &i_alpha,
                          A,
                          AType,
                          m,
                          B,
                          BType,
                          m,
                          &i_beta,
                          C,
                          CType,
                          n,
                          ComputeType,
                          static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("Error: cublas error.\n");
        exit(0);
    }
    hipDeviceSynchronize();
}

void * FastllmCudaMalloc(size_t size) {
    void * ret;
    hipMalloc(&ret, size);
    return ret;
}

void FastllmCudaCopyFromHostToDevice(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void FastllmCudaCopyFromDeviceToHost(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void FastllmCudaFree(void *ret) {
    hipFree(ret);
}